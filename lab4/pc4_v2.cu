#include "hip/hip_runtime.h"
 /*
 Author: Deren Kong (Kong.325)
 CSE5441  lab4
 Description: a simple openmp program to accomplish producer and consumer function
 compile: qsub -I -l walltime=00:59:00 -l nodes=1:gpus=1,mem=4gb -A PAS1421
	  sh ./fire_cuda_v2.sh
 Submitted on : 11/27/2018
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include "pc4_v2.h"
#define EMPTY 9999 
__device__ u_int16_t transformA(u_int16_t input_val);
__device__ u_int16_t transformB(u_int16_t input_val);
__device__ u_int16_t transformC(u_int16_t input_val);
__device__ u_int16_t transformD(u_int16_t input_val);
__global__ void transform(char* cmd, u_int16_t *pre, u_int16_t *aft) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	//for (int i = 0; i < 10; i++) {
	if (cmd[id] == 'A') {
		aft[id] = transformA(pre[id]);
	}
	if (cmd[id] == 'B') {
		aft[id] = transformB(pre[id]);
	}
	if (cmd[id] == 'C') {
		aft[id] = transformC(pre[id]);
	}
	if (cmd[id] == 'D') {
		aft[id] = transformD(pre[id]);
	}
}
__global__ void transform_print(char* cmd, u_int16_t *pre, u_int16_t *aft) {
	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (cmd[id] == 'A') {
		aft[id] = transformA(pre[id]);
	}
	if (cmd[id] == 'B') {
		aft[id] = transformB(pre[id]);
	}
	if (cmd[id] == 'C') {
		aft[id] = transformC(pre[id]);
	}
	if (cmd[id] == 'D') {
		aft[id] = transformD(pre[id]);
	}

	printf("Q:%d %c %hd %hd\n",id, cmd[id], pre[id], aft[id]);
}
queue *init_queue() {	
	queue * queue_list;
	queue_list = (queue*) malloc (sizeof(queue));
	for (int i = 0; i < N; i++) {
		queue_list->work_queue[i].cmd = ' ';
		queue_list->work_queue[i].key = EMPTY;
	}
	queue_list->empty = true;
	queue_list->full = false;
	queue_list->head = 0;
	queue_list->tail = 0;
	return queue_list;
}

void destroy_queue(queue *q) {
	free(q);
}

void add_queue(queue *q, work_entry element) {
	q->work_queue[q->tail] = element;
	q->tail++;
	if (q->tail == N) {
		q->tail = 0;
	}
	if (q->tail == q->head) {
		q->full = true;
	}	
	q->empty = false;
}

void del_queue(queue *q, work_entry *out) {
	*out = q->work_queue[q->head];
	q->head++;
	if (q->head == N) {
		q->head = 0;
	}
	if (q->head == q->tail) {
		q->empty = true;	
	}
	q->full = false;
}

// producer are able to transform each key and fill
//that into work queue, when the queue has been filled, it will stop and wait for 
//consumer to clean it. 
void producer(queue *queue_list_a, queue* queue_list_b, queue* queue_list_c,
		queue* queue_list_d, int *run_time, double *run_time2) {
	char buffer[20];
	char temp_key[10];
	char cmd = ' ';
	//read each line to buffer
	while(fgets(buffer, 20, stdin) != NULL) {
	sscanf(buffer, "%c  %s", &cmd, temp_key);
	u_int16_t key = (u_int16_t)(atoi(temp_key));
	work_entry buf;
// this version use 4 queue_list to collect A B C D seperately 
	if (cmd == 'X') {
    		break;
	}
	if (cmd == 'A' && key <= 1000) {
		buf.cmd = cmd;
		buf.key = key;
		add_queue(queue_list_a, buf);
	}
	if (cmd == 'B' && key <= 1000) {
		buf.cmd = cmd;
		buf.key = key;
		add_queue(queue_list_b, buf);
	}
	if (cmd == 'C' && key <= 1000) {
		buf.cmd = cmd;
		buf.key = key;
		add_queue(queue_list_c, buf);
	}
	if (cmd == 'D' && key <= 1000) {
		buf.cmd = cmd;
		buf.key = key;
		add_queue(queue_list_d, buf);
	}
		}
	printf ("finish reading!\n");
	int num_blocks = 1;
	int num_th_per_blk_a = queue_list_a->tail;
	int num_th_per_blk_b = queue_list_b->tail;
	int num_th_per_blk_c = queue_list_c->tail;
	int num_th_per_blk_d = queue_list_d->tail;
	int num_th_per_blk = queue_list_a->tail + queue_list_b->tail + queue_list_c->tail + queue_list_d->tail;
	int old_num_th_per_blk = -1;
	if (N >= 2048) {
		num_blocks = 10;
		printf("original is %d ",num_th_per_blk);
		old_num_th_per_blk = num_th_per_blk;
		num_th_per_blk = ceil((double)num_th_per_blk / 10.0);
		printf("now is %d\n", num_th_per_blk);
	}
	char * pre_cmd_h, *pre_cmd_d;;
	u_int16_t *pre_key_h, *aft_key_h, *pre_key_d, *aft_key_d;	
	size_t memSize_cmd = num_blocks*num_th_per_blk*sizeof(char);
	size_t memSize_key = num_blocks*num_th_per_blk*sizeof(u_int16_t);
	pre_cmd_h = (char*) malloc(memSize_cmd);
	pre_key_h = (u_int16_t*) malloc(memSize_key);
	aft_key_h = (u_int16_t*)malloc(memSize_key);
	for (int i = 0; i < num_th_per_blk_a; i++) {
		pre_cmd_h[i] = queue_list_a->work_queue[i].cmd;
		pre_key_h[i] = queue_list_a->work_queue[i].key;
		//printf("%c %hd\n", pre_cmd_h[i], pre_key_h[i]);
	}
	for (int i = 0; i < num_th_per_blk_b; i++) {
		pre_cmd_h[num_th_per_blk_a + i] = queue_list_b->work_queue[i].cmd;
		pre_key_h[num_th_per_blk_a + i] = queue_list_b->work_queue[i].key;
		//printf("%c %hd\n", pre_cmd_h[num_th_per_blk_a+ i], pre_key_h[num_th_per_blk_a+ i]);
	}
	for (int i = 0; i <  num_th_per_blk_c; i++) {
		pre_cmd_h[num_th_per_blk_a + num_th_per_blk_b + i] = queue_list_c->work_queue[i].cmd;
		pre_key_h[num_th_per_blk_a + num_th_per_blk_b + i] = queue_list_c->work_queue[i].key;
		//printf("%c %hd\n", pre_cmd_h[num_th_per_blk_a + num_th_per_blk_b + i], pre_key_h[num_th_per_blk_a + num_th_per_blk_b + i]);
	}
	for (int i = 0; i < num_th_per_blk_d; i++) {
		pre_cmd_h[num_th_per_blk_a + num_th_per_blk_b + num_th_per_blk_c + i] = queue_list_d->work_queue[i].cmd;
		pre_key_h[num_th_per_blk_a + num_th_per_blk_b + num_th_per_blk_c + i] = queue_list_d->work_queue[i].key;
		//printf("%c %hd\n", pre_cmd_h[num_th_per_blk_a + num_th_per_blk_b + num_th_per_blk_c +i], pre_key_h[num_th_per_blk_a + num_th_per_blk_b + num_th_per_blk_c +i]);
	}
	clock_t t;
	t = clock();
	time_t timer;
	timer = time(NULL);

	hipMalloc((void**)&pre_cmd_d, memSize_cmd);
	hipMalloc((void**)&pre_key_d, memSize_key);
	hipMalloc((void**)&aft_key_d, memSize_key);
	hipMemcpy(pre_cmd_d, pre_cmd_h, memSize_cmd, hipMemcpyHostToDevice);
	hipMemcpy(pre_key_d, pre_key_h, memSize_key, hipMemcpyHostToDevice);
	printf("launch kernel\n");
	dim3 dimGrid(num_blocks);
	dim3 dimBlock(num_th_per_blk);
	transform <<< dimGrid, dimBlock >>>(pre_cmd_d, pre_key_d, aft_key_d);

	printf("finish transform\n");
	hipMemcpy(pre_cmd_h, pre_cmd_d, memSize_cmd, hipMemcpyDeviceToHost);
	hipMemcpy(pre_key_h, pre_key_d, memSize_key, hipMemcpyDeviceToHost);

	hipMemcpy(aft_key_h, aft_key_d, memSize_key, hipMemcpyDeviceToHost);
	hipFree(aft_key_d);
	hipFree(pre_cmd_d);
	hipFree(aft_key_d);
	printf ("finish producer!\n");	
	consumer(pre_cmd_h, aft_key_h, num_th_per_blk);
	time_t cur = time(NULL);
        *run_time2 += difftime(cur,timer);
        t = clock() - t;
        *run_time += t;
	}

// consumer are able to transform the encoded key 
// and print result with stdout
void consumer(char* cmd, u_int16_t* key, int size) {

	char* cmd_d;
	u_int16_t *key_d, *aft_key_h, *aft_key_d;
	int num_blocks = 1;
	int num_th_per_blk = size;
	if (N >= 2048) {
		num_blocks = 10;
	}
	size_t memSize_cmd = num_blocks*num_th_per_blk*sizeof(char);
	size_t memSize_key = num_blocks*num_th_per_blk*sizeof(u_int16_t);
	hipMalloc((void**) &cmd_d, memSize_cmd);
	hipMalloc((void**) &key_d, memSize_key);
	hipMalloc((void**) &aft_key_d, memSize_key);
	hipMemcpy(key_d, key, memSize_key, hipMemcpyHostToDevice);
	hipMemcpy(cmd_d, cmd, memSize_cmd, hipMemcpyHostToDevice);

	dim3 dimGrid(num_blocks);
	dim3 dimBlock(num_th_per_blk);
	transform_print <<< dimGrid, dimBlock >>>(cmd_d, key_d, aft_key_d);

	printf("consumer cuda work done!\n");
	hipMemcpy(aft_key_h, aft_key_d, memSize_key, hipMemcpyDeviceToHost);
	hipFree(cmd_d);
	hipFree(key_d);
	hipFree(aft_key_d);
	}

int main(int argc, char* argv[]) {
	queue* queue_list_a = init_queue();
	queue* queue_list_b = init_queue();
	queue* queue_list_c = init_queue();
	queue* queue_list_d = init_queue();
	int run_time = 0;
	double run_time2 = 0.0;
	producer(queue_list_a, queue_list_b, queue_list_c, queue_list_d, &run_time, &run_time2);
	printf("The total runtime of producer and consumer is %d clicks (%f seconds).\n", (run_time),((float )run_time)/(CLOCKS_PER_SEC));
	printf("The total runtime2 is %lf seconds.\n", (run_time2));
	free(queue_list_a);
	free(queue_list_b);
	free(queue_list_c);
	free(queue_list_d);
	return 0;
}



