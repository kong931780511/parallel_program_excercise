#include "hip/hip_runtime.h"
 /*
 Author: Deren Kong (Kong.325)
 CSE5441  lab4
 Description: a simple cuda program to accomplish producer and consumer function
 Compile:
	qsub -I -l walltime=00:59:00 -l nodes=1:gpus=1,mem=4gb -A PAS1421
        sh ./fire_cuda.sh
 Submitted on : 11/27/2018
*/

#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <time.h>
#include <string.h>
#include <time.h>
#include "pc4.h"
#define EMPTY 9999 
__device__ u_int16_t transformA(u_int16_t input_val);
__device__ u_int16_t transformB(u_int16_t input_val);
__device__ u_int16_t transformC(u_int16_t input_val);
__device__ u_int16_t transformD(u_int16_t input_val);
//kernel for producer
__global__ void transform(char* cmd, u_int16_t *pre, u_int16_t *aft) {
	int id = threadIdx.x;
	if (cmd[id] == 'A') {
		aft[id] = transformA(pre[id]);
	}
	if (cmd[id] == 'B') {
		aft[id] = transformB(pre[id]);
	}
	if (cmd[id] == 'C') {
		aft[id] = transformC(pre[id]);
	}
	if (cmd[id] == 'D') {
		aft[id] = transformD(pre[id]);
	}
}
//kernel for consumer
__global__ void transform_print(char* cmd, u_int16_t *pre, u_int16_t *aft) {
	int id = threadIdx.x;
	if (cmd[id] == 'A') {
		aft[id] = transformA(pre[id]);
	}
	if (cmd[id] == 'B') {
		aft[id] = transformB(pre[id]);
	}
	if (cmd[id] == 'C') {
		aft[id] = transformC(pre[id]);
	}
	if (cmd[id] == 'D') {
		aft[id] = transformD(pre[id]);
	}
	printf("Q:%d %c %hd %hd\n",id, cmd[id], pre[id], aft[id]);
}
queue *init_queue() {	
	queue * queue_list;
	queue_list = (queue*) malloc (sizeof(queue));
	for (int i = 0; i < N; i++) {
		queue_list->work_queue[i].cmd = ' ';
		queue_list->work_queue[i].key = EMPTY;
	}
	queue_list->empty = true;
	queue_list->full = false;
	queue_list->head = 0;
	queue_list->tail = 0;
	return queue_list;
}

void destroy_queue(queue *q) {
	free(q);
}

void add_queue(queue *q, work_entry element) {
	q->work_queue[q->tail] = element;
	q->tail++;
	if (q->tail == N) {
		q->tail = 0;
	}
	if (q->tail == q->head) {
		q->full = true;
	}	
	q->empty = false;
}

void del_queue(queue *q, work_entry *out) {
	*out = q->work_queue[q->head];
	q->head++;
	if (q->head == N) {
		q->head = 0;
	}
	if (q->head == q->tail) {
		q->empty = true;	
	}
	q->full = false;
}

//producer input all the data to cuda and when it finished,
//just fire consumer
void producer(queue *queue_list, int *run_time, double *run_time2) {
	char buffer[20];
	char temp_key[10];
	char cmd = ' ';
	while(fgets(buffer, 20, stdin) != NULL) {
	sscanf(buffer, "%c  %s", &cmd, temp_key);
	u_int16_t key = (u_int16_t)(atoi(temp_key));
	work_entry buf;
	if (cmd == 'X') {
     		break;
	}
	if (cmd != 'A' && cmd != 'B' && cmd != 'C' && cmd != 'D') {
		continue;
	}
	if (key <= 1000) {
		buf.cmd = cmd;
		buf.key = key;
		add_queue(queue_list, buf);
	}
	}
	printf ("finish reading!\n");
	int num_blocks = 1;
	int num_th_per_blk = queue_list->tail;
// if N is larger than 2048, then just set numGrid = 10 and numBlock = N/10
// if N / 10 has remainer, then just padding it.
	if (N >= 2048) {
                num_blocks = 10;
                num_th_per_blk = ceil((double)num_th_per_blk / 10.0);
        }
	clock_t t;
        t = clock();
        time_t timer;
        timer = time(NULL);
	char * pre_cmd_h, *pre_cmd_d;
	u_int16_t *pre_key_h, *aft_key_h, *pre_key_d, *aft_key_d;
	size_t memSize_cmd = num_blocks*num_th_per_blk*sizeof(char);
	size_t memSize_key = num_blocks*num_th_per_blk*sizeof(u_int16_t);
	pre_cmd_h = (char*) malloc(memSize_cmd);
	pre_key_h = (u_int16_t*) malloc(memSize_key);
	aft_key_h = (u_int16_t*)malloc(memSize_key);
// save all keys and cmds to arrays
	for (int i = 0; i < num_th_per_blk; i++) {
		pre_cmd_h[i] = queue_list->work_queue[i].cmd;
		pre_key_h[i] = queue_list->work_queue[i].key;
	}
	hipMalloc((void**)&pre_cmd_d, memSize_cmd);
	hipMalloc((void**)&pre_key_d, memSize_key);
	hipMalloc((void**)&aft_key_d, memSize_key);
	hipMemcpy(pre_cmd_d, pre_cmd_h, memSize_cmd, hipMemcpyHostToDevice);
	hipMemcpy(pre_key_d, pre_key_h, memSize_key, hipMemcpyHostToDevice);
	dim3 dimGrid(num_blocks);
	dim3 dimBlock(num_th_per_blk);
	transform <<< dimGrid, dimBlock >>>(pre_cmd_d, pre_key_d, aft_key_d);
	hipMemcpy(pre_cmd_h, pre_cmd_d, memSize_cmd, hipMemcpyDeviceToHost);
	hipMemcpy(pre_key_h, pre_key_d, memSize_key, hipMemcpyDeviceToHost);
	hipMemcpy(aft_key_h, aft_key_d, memSize_key, hipMemcpyDeviceToHost);
	hipFree(pre_cmd_d);
	hipFree(pre_key_d);
	hipFree(aft_key_d);

	consumer(pre_cmd_h, aft_key_h, num_th_per_blk);
	time_t cur = time(NULL);
        *(run_time2) += difftime(cur,timer);
        t = clock() - t;
        *(run_time) += t;
	}

// consumer are able to transform the encoded key 
// and print result with stdout
void consumer(char* cmd, u_int16_t* key, int size) {
	char* cmd_d;
	u_int16_t *key_d, *aft_key_h, *aft_key_d;
	int num_blocks = 1;
	int num_th_per_blk = size;
// same as consumer, just used when dimGrid and dimBolck changed in producer
	if (N >= 2048) {
                num_blocks = 10;
        }
	size_t memSize_cmd = num_blocks*num_th_per_blk*sizeof(char);
	size_t memSize_key = num_blocks*num_th_per_blk*sizeof(u_int16_t);
	hipMalloc((void**) &cmd_d, memSize_cmd);
	hipMalloc((void**) &key_d, memSize_key);
	hipMalloc((void**) &aft_key_d, memSize_key);
	hipMemcpy(key_d, key, memSize_key, hipMemcpyHostToDevice);
	hipMemcpy(cmd_d, cmd, memSize_cmd, hipMemcpyHostToDevice);
	dim3 dimGrid(num_blocks);
	dim3 dimBlock(num_th_per_blk);
	transform_print <<< dimGrid, dimBlock >>>(cmd_d, key_d, aft_key_d);
	hipFree(cmd_d);
	hipFree(key_d);
	hipFree(aft_key_d);
	}

int main(int argc, char* argv[]) {
	queue * queue_list = init_queue();
        if (queue_list == NULL) {
        	printf("Failed to init!\n");
        }
	int run_time = 0;
	double run_time2 = 0.0;
	producer(queue_list, &run_time, &run_time2);
	printf("The total runtime of producer and consumer is %d clicks (%f seconds).\n", (run_time),((float )run_time)/(CLOCKS_PER_SEC));
	printf("The total runtime2 is %lf seconds.\n", (run_time2));
	destroy_queue(queue_list);
	return 0;
}



